
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

/*! BiasAdd <T = float32, Device = CUDA> */

template <typename T>
__global__ void _BiasAdd_NCHW(
    const int               count,
    const int               dim,
    const int               inner_dim,
    const T*                bias,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] += bias[(idx / inner_dim) % dim];
    }
}

template <typename T>
__global__ void _BiasAdd_NHWC(
    const int               count,
    const int               dim,
    const int               inner_dim,
    const T*                bias,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] += bias[idx % dim];
    }
}

template<> void BiasAdd<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               dim,
    const int               inner_dim,
    const string&           data_format,
    const float*            bias,
    const float*            bias_multiplier,
    float*                  y,
    CUDAContext*            ctx) {
    if (data_format == "NCHW") {
        _BiasAdd_NCHW<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (count, dim, inner_dim, bias, y);
    } else if (data_format == "NHWC") {
        _BiasAdd_NHWC<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (count, dim, inner_dim, bias, y);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA