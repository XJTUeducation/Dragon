
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

/*! MaximumE <T = float32, Device = CUDA> */

template <typename T>
__global__ void _MaximumE(
    const int               count,
    const T*                x1,
    const T*                x2,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = max(x1[idx], x2[idx]);
    }
}

template <> void MaximumE<float, CUDAContext>(
    const int               count,
    const float*            x1,
    const float*            x2,
    float*                  y,
    CUDAContext*            ctx) {
    _MaximumE<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (count, x1, x2, y);
}

/*! MaximumB <T = float32, Device = CUDA> */

template <typename T>
__global__ void _MaximumB(
    const int               count,
    const T*                x1,
    const T                 x2,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = max(x1[idx], x2);
    }
}

template <> void MaximumB<float, CUDAContext>(
    const int               count,
    const float*            x1,
    const float             x2,
    float*                  y,
    CUDAContext*            ctx) {
    _MaximumB<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (count, x1, x2, y);
}

/*! MaximumEGrad <T = float32, Device = CUDA> */

template <typename T>
__global__ void _MaximumEGrad(
    const int               count,
    const T*                x1,
    const T*                x2,
    const T*                dy,
    T*                      dx1,
    T*                      dx2) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const bool dy_to_dx1 = x1[idx] > x2[idx];
        dx1[idx] = dy_to_dx1 ? dy[idx] : 0;
        dx2[idx] = dy_to_dx1 ? 0 : dy[idx];
    }
}

template <> void MaximumEGrad<float, CUDAContext>(
    const int               count,
    const float*            x1,
    const float*            x2,
    const float*            dy,
    float*                  dx1,
    float*                  dx2,
    CUDAContext*            ctx) {
    _MaximumEGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (count, x1, x2, dy, dx1, dx2);
}

/*! MaximumBGrad <T = float32, Device = CUDA> */

template <typename T>
__global__ void _MaximumBGrad(
    const int               count,
    const T*                x1,
    const T                 x2,
    const T*                dy,
    T*                      dx1) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        dx1[idx] = (x1[idx] > x2) ? dy[idx] : 0;
    }
}

template <> void MaximumBGrad<float, CUDAContext>(
    const int               count,
    const float*            x1,
    const float             x2,
    const float*            dy,
    float*                  dx1,
 /* float*                  dx2, */
    CUDAContext*            ctx) {
    _MaximumBGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (count, x1, x2, dy, dx1);
}

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA