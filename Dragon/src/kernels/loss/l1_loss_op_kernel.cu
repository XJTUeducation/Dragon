
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

/*! AbsGrad <T = float32, Device = CUDA> */

template <typename T>
__global__ void _AbsGrad(
    const int               count,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
       const T val = dy[idx];
       //  val > 0: 1 | val == 0: 0 | val < 0: -1
       dx[idx] = (val > T(0)) - (val < T(0));
    }
}

template<> void AbsGrad<float, CUDAContext>(
    const int               count,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _AbsGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (count, dy, dx);
}

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA