
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/op_kernel.h"
#include "utils/math_functions.h"

namespace dragon {

namespace kernel {

/*! PRelu <T = float32, Device = CUDA> */

template <typename T>
__global__ void _PRelu(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                x,
    const T*                w,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = (x[idx] > 0) * x[idx] +
            (x[idx] < 0) * x[idx] * w[0];
    }
}

template <typename T>
__global__ void _PReluNCHW(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                x,
    const T*                w,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = (idx / dim) % channels;
        y[idx] = (x[idx] > 0) * x[idx] +
            (x[idx] < 0) * x[idx] * w[c];
    }
}

template <typename T>
__global__ void _PReluNHWC(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                x,
    const T*                w,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % channels;
        y[idx] = (x[idx] > 0) * x[idx] +
            (x[idx] < 0) * x[idx] * w[c];
    }
}

template<> void PRelu<float, CUDAContext>(
    const int               count,
    const int               channels,
    const int               dim,
    const bool              channel_shared,
    const string&           data_format,
    const float*            x,
    const float*            w,
    float*                  y,
    CUDAContext*            ctx) {
    if (channel_shared) {
        _PRelu<float> 
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (count, channels, dim, x, w, y);
    } else {
        if (data_format == "NCHW") {
            _PReluNCHW<float>
                << < CUDA_BLOCKS(count), CUDA_THREADS,
                     0, ctx->cuda_stream() >> >
                (count, channels, dim, x, w, y);
        } else if (data_format == "NHWC") {
            _PReluNHWC<float>
                << < CUDA_BLOCKS(count), CUDA_THREADS,
                     0, ctx->cuda_stream() >> >
                (count, channels, dim, x, w, y);
        } else LOG(FATAL) << "Unknown data format: " << data_format;
    }
}

/*! PReluGrad <T = float32, Device = CUDA> */

template <typename T>
__global__ void _PReluGrad(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                dy,
    const T*                x,
    const T*                w,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * (
            (x[idx] > 0) + (x[idx] <= 0) * w[0]
        );
    }
}

template <typename T>
__global__ void _PReluGradNCHW(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                dy,
    const T*                x,
    const T*                w,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = (idx / dim) % channels;
        dx[idx] = dy[idx] * (
            (x[idx] > 0) + (x[idx] <= 0) * w[c]
        );
    }
}

template <typename T>
__global__ void _PReluGradNHWC(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                dy,
    const T*                x,
    const T*                w,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % channels;
        dx[idx] = dy[idx] * ((x[idx] > 0) + (x[idx] <= 0) * w[c]);
    }
}

template<> void PReluGrad<float, CUDAContext>(
    const int               count,
    const int               channels,
    const int               dim,
    const bool              channel_shared,
    const string&           data_format,
    const float*            dy,
    const float*            x,
    const float*            w,
    float*                  dx,
    CUDAContext*            ctx) {
    if (channel_shared) {
        _PReluGrad<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (count, channels, dim, dy, x, w, dx);
    } else {
        if (data_format == "NCHW") {
            _PReluGradNCHW<float>
                << < CUDA_BLOCKS(count), CUDA_THREADS,
                     0, ctx->cuda_stream() >> >
                (count, channels, dim, dy, x, w, dx);
        } else if (data_format == "NHWC") {
            _PReluGradNHWC<float>
                << < CUDA_BLOCKS(count), CUDA_THREADS,
                     0, ctx->cuda_stream() >> >
                (count, channels, dim, dy, x, w, dx);
        } else LOG(FATAL) << "Unknown data format: " << data_format;
    }
}

/*! PReluWGrad <T = float32, Device = CUDA> */

template <typename T>
__global__ void _PReluWGradBcast(
    const int               count,
    const int               rows,
    const int               row_offset,
    const T*                dy,
    const T*                x,
    T*                      bcast_dw) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        bcast_dw[idx] = dy[idx] * x[idx] * (x[idx] <= 0);
        for (int n = 1; n < rows; n++) {
            const int cur_idx = idx + n * row_offset;
            bcast_dw[idx] +=
                dy[cur_idx] * x[cur_idx] * (x[cur_idx] <= 0);
        }
    }
}

template<> void PReluWGrad<float, CUDAContext>(
    const int               rows,
    const int               row_offset,
    const int               channels,
    const int               dim,
    const bool              channel_shared,
    const string&           data_format,
    const float*            dy,
    const float*            x,
    const float*            multiplier,
    float*                  bcast_dw,
    float*                  dw,
    CUDAContext*            ctx) {
    const int cdim = channels * dim;
    _PReluWGradBcast<float>
        << < CUDA_BLOCKS(cdim), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (cdim, rows, row_offset, dy, x, bcast_dw);
    if (channel_shared) {
        float w_sum;
        math::Dot<float, CUDAContext>(channels * dim,
            bcast_dw, multiplier, &w_sum, ctx);
        math::AddScalar<float, CUDAContext>(1, w_sum, dw, ctx);
    } else {
        if (data_format == "NCHW") {
            math::Gemv<float, CUDAContext>(
                CblasNoTrans, channels, dim,
                    1.f, bcast_dw, multiplier,
                        1.f, dw, ctx);
        } else if (data_format == "NHWC") {
            math::Gemv<float, CUDAContext>(
                CblasTrans, dim, channels,
                    1.f, bcast_dw, multiplier, 
                        1.f, dw, ctx);
        } else LOG(FATAL) << "Unknown data format: " << data_format;
    }
}

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA